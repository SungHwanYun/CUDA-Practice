#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

/*
 * A simple example of using an array of structures to store data on the device.
 * This example is used to study the impact on performance of data layout on the
 * GPU.
 *
 * AoS: one contiguous 64-bit read to get x and y (up to 300 cycles)
 */

#define LEN 1<<22
#define CHECK(call) \
{ \
    const hipError_t error = call;\
    if (error != hipSuccess) { \
        printf("[device] Error: %s %d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(-10*error); \
    } \
}
struct innerStruct {
    float x;
    float y;
};

struct innerArray {
    float x[LEN];
    float y[LEN];
};

void initialInnerStruct(innerStruct* ip, int size) {
    for (int i = 0; i < size; i++)     {
        ip[i].x = (float)(rand() & 0xFF) / 100.0f;
        ip[i].y = (float)(rand() & 0xFF) / 100.0f;
    }
}

void testInnerStructHost(innerStruct* A, innerStruct* C, const int n) {
    for (int idx = 0; idx < n; idx++) {
        C[idx].x = A[idx].x + 10.f;
        C[idx].y = A[idx].y + 20.f;
    }
}

void checkInnerStruct(innerStruct* hostRef, innerStruct* gpuRef, const int N) {
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i].x - gpuRef[i].x) > epsilon) {
            match = 0;
            printf("[host] different on %dth element: host %f gpu %f\n", 
                i, hostRef[i].x, gpuRef[i].x);
            break;
        }
        if (abs(hostRef[i].y - gpuRef[i].y) > epsilon) {
            match = 0;
            printf("[host] different on %dth element: host %f gpu %f\n", 
                i, hostRef[i].y, gpuRef[i].y);
            break;
        }
    }
    if (!match)  printf("[host] Arrays do not match.\n\n");
}

__global__ void testInnerStruct(innerStruct* data, innerStruct* result, const int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        innerStruct tmp = data[i];
        tmp.x += 10.f;
        tmp.y += 20.f;
        result[i] = tmp;
    }
}

__global__ void warmup(innerStruct* data, innerStruct* result, const int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        innerStruct tmp = data[i];
        tmp.x += 10.f;
        tmp.y += 20.f;
        result[i] = tmp;
    }
}

int main(int argc, char** argv) {
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[host] %s test struct of array at ", argv[0]);
    printf("device %d: %s \n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // allocate host memory
    int nElem = LEN;
    size_t nBytes = nElem * sizeof(innerStruct);
    innerStruct* h_A = (innerStruct*)malloc(nBytes);
    innerStruct* hostRef = (innerStruct*)malloc(nBytes);
    innerStruct* gpuRef = (innerStruct*)malloc(nBytes);

    // initialize host array
    initialInnerStruct(h_A, nElem);
    testInnerStructHost(h_A, hostRef, nElem);

    // allocate device memory
    innerStruct* d_A, * d_C;
    CHECK(hipMalloc((innerStruct**)&d_A, nBytes));
    CHECK(hipMalloc((innerStruct**)&d_C, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

    // set up offset for summaryAU: It is blocksize not offset. Thanks.CZ
    int blocksize = 128;

    if (argc > 1) blocksize = atoi(argv[1]);

    // execution configuration
    dim3 block(blocksize, 1);
    dim3 grid((nElem + block.x - 1) / block.x, 1);

    // kernel 1: warmup
    warmup << <grid, block >> > (d_A, d_C, nElem);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkInnerStruct(hostRef, gpuRef, nElem);
    CHECK(hipGetLastError());

    // kernel 2: testInnerStruct
    testInnerStruct << <grid, block >> > (d_A, d_C, nElem);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkInnerStruct(hostRef, gpuRef, nElem);
    CHECK(hipGetLastError());

    // free memories both host and device
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_C));
    free(h_A);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}

/*
output:
c:\coding\Cuda\x64\Debug>nvprof ./Cuda.exe
==25996== NVPROF is profiling process 25996, command: ./Cuda.exe
[host] ./Cuda.exe test struct of array at device 0: NVIDIA GeForce MX450
==25996== Profiling application: ./Cuda.exe
==25996== Warning: 43 API trace records have same start and end timestamps.
This can happen because of short execution duration of CUDA APIs and low timer resolution on the underlying operating system.
==25996== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   60.09%  21.214ms         2  10.607ms  10.543ms  10.672ms  [CUDA memcpy DtoH]
                   30.84%  10.889ms         1  10.889ms  10.889ms  10.889ms  [CUDA memcpy HtoD]
                    4.54%  1.6020ms         1  1.6020ms  1.6020ms  1.6020ms  warmup(innerStruct*, innerStruct*, int)
                    4.54%  1.6016ms         1  1.6016ms  1.6016ms  1.6016ms  testInnerStruct(innerStruct*, innerStruct*, int)
      API calls:   52.89%  85.895ms         1  85.895ms  85.895ms  85.895ms  hipSetDevice
                   21.57%  35.025ms         3  11.675ms  10.974ms  12.948ms  hipMemcpy
                   20.07%  32.601ms         1  32.601ms  32.601ms  32.601ms  hipDeviceReset
                    2.29%  3.7206ms         2  1.8603ms  87.700us  3.6329ms  hipLaunchKernel
                    2.10%  3.4123ms         2  1.7062ms  1.6864ms  1.7259ms  hipDeviceSynchronize
                    0.77%  1.2541ms         2  627.05us  508.40us  745.70us  hipFree
                    0.26%  420.00us         2  210.00us  130.10us  289.90us  hipMalloc
                    0.02%  31.000us         1  31.000us  31.000us  31.000us  hipLibraryUnload
                    0.02%  24.700us         1  24.700us  24.700us  24.700us  hipDeviceTotalMem
                    0.01%  17.200us       114     150ns       0ns  2.1000us  hipDeviceGetAttribute
                    0.00%  2.7000us         1  2.7000us  2.7000us  2.7000us  hipGetDeviceProperties
                    0.00%  2.3000us         2  1.1500us  1.1000us  1.2000us  hipGetLastError
                    0.00%  2.1000us         3     700ns     100ns  1.7000us  hipGetDeviceCount
                    0.00%  1.9000us         1  1.9000us  1.9000us  1.9000us  cuModuleGetLoadingMode
                    0.00%     900ns         1     900ns     900ns     900ns  hipDeviceGetName
                    0.00%     700ns         2     350ns     100ns     600ns  hipDeviceGet
                    0.00%     300ns         1     300ns     300ns     300ns  cuDeviceGetLuid
                    0.00%     200ns         1     200ns     200ns     200ns  hipDeviceGetUuid
*/
