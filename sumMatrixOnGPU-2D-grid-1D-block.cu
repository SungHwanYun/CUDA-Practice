#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<string.h>

// error check
#define CHECK(call) \
{ \
    const hipError_t error = call;\
    if (error != hipSuccess) { \
        printf("[device] Error: %s %d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(-10*error); \
    } \
}

// Initialize the data pointed to by the pointer ip.
void initialInt(int* ip, const int N) {
    for (int i = 0; i < N; i++) {
        ip[i] = i;
    }
}

// Check if the computation results of the CPU and GPU are the same
void checkResult(float* hostRef, float* gpuRef, const int N) {
    double epsilon = 1.0e-8;
    bool match = 1;
    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("[host] Arrays do not match!\n");
            printf("[host] host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if (match) printf("[host] Arrays match.\n\n");
}

void sumMatrixOnHost(float* A, float* B, float* C, const int nx, const int ny) {
    float* ia = A, * ib = B, * ic = C;
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx; ib += nx; ic += nx;
    }
}

// Initialize the data pointed to by the pointer ip.
#include<time.h>
void initialData(float* ip, const int N) {
    // generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0; i < N; i++) {
        ip[i] = (float)(rand() & 0xff) / 10.0f;
    }
}

__global__ void sumMatrixOnGPU2D2D(float* A, float* B, float* C, const int nx, const int ny) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny) {
        C[idx] = A[idx] + B[idx];
    }
}
__global__ void sumMatrixOnGPU1D1D(float* A, float* B, float* C, const int nx, const int ny) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix < nx) {
        for (int iy = 0; iy < ny; iy++) {
            unsigned int idx = iy * nx + ix;
            C[idx] = A[idx] + B[idx];
        }
    }
}
__global__ void sumMatrixOnGPU2D1D(float* A, float* B, float* C, const int nx, const int ny) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny) {
        C[idx] = A[idx] + B[idx];
    }
}
int main(int argc, char **argv) {
    printf("[host] %s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[host] Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int nx = 1 << 14;
    int ny = 1 << 14;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("[host] Matrix size : nx %d, ny %d\n", nx, ny);

    // malloc host memory
    float* h_A, * h_B, * hostRef, * gpuRef;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);

    // initialize data at host side
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);

    // malloc device global memory
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, nBytes);
    hipMalloc((void**)&d_B, nBytes);
    hipMalloc((void**)&d_C, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // set up execution configuration
    int dimx = 32;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, ny);

    // invoke the kernel
    sumMatrixOnGPU2D1D << <grid, block >> > (d_A, d_B, d_C, nx, ny);
    hipDeviceSynchronize();

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // check device results
    checkResult(hostRef, gpuRef, nxy);
    
    // free host and device memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(hostRef); free(gpuRef);

    // reset device
    hipDeviceReset();
}

/*
output:
C:\coding\Cuda\x64\Debug>nvprof ./Cuda.exe
[host] ./Cuda.exe Starting...
==11636== NVPROF is profiling process 11636, command: ./Cuda.exe
[host] Using Device 0: NVIDIA GeForce MX450
[host] Matrix size : nx 16384, ny 16384
[host] Arrays match.

==11636== Profiling application: ./Cuda.exe
==11636== Warning: 37 API trace records have same start and end timestamps.
This can happen because of short execution duration of CUDA APIs and low timer resolution on the underlying operating system.
==11636== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   49.38%  739.83ms         2  369.92ms  363.57ms  376.26ms  [CUDA memcpy HtoD]
                   27.81%  416.65ms         1  416.65ms  416.65ms  416.65ms  [CUDA memcpy DtoH]
                   22.81%  341.81ms         1  341.81ms  341.81ms  341.81ms  sumMatrixOnGPU2D1D(float*, float*, float*, int, int)
      API calls:   59.22%  1.43075s         3  476.92ms  363.61ms  650.09ms  hipMemcpy
                   15.06%  363.82ms         3  121.27ms  2.2682ms  341.71ms  hipMalloc
                   14.15%  341.86ms         1  341.86ms  341.86ms  341.86ms  hipDeviceSynchronize
                    6.85%  165.54ms         3  55.181ms  10.570ms  134.70ms  hipFree
                    3.08%  74.413ms         1  74.413ms  74.413ms  74.413ms  hipSetDevice
                    1.58%  38.265ms         1  38.265ms  38.265ms  38.265ms  hipDeviceReset
                    0.06%  1.3330ms         1  1.3330ms  1.3330ms  1.3330ms  hipLaunchKernel
                    0.00%  19.200us       114     168ns       0ns  2.8000us  hipDeviceGetAttribute
                    0.00%  15.600us         1  15.600us  15.600us  15.600us  hipLibraryUnload
                    0.00%  4.9000us         2  2.4500us       0ns  4.9000us  hipDeviceGet
                    0.00%  3.5000us         1  3.5000us  3.5000us  3.5000us  hipGetDeviceProperties
                    0.00%  2.3000us         3     766ns     100ns  1.9000us  hipGetDeviceCount
                    0.00%  2.0000us         1  2.0000us  2.0000us  2.0000us  cuModuleGetLoadingMode
                    0.00%  2.0000us         1  2.0000us  2.0000us  2.0000us  hipDeviceTotalMem
                    0.00%     900ns         1     900ns     900ns     900ns  hipDeviceGetName
                    0.00%     200ns         1     200ns     200ns     200ns  cuDeviceGetLuid
                    0.00%     100ns         1     100ns     100ns     100ns  hipDeviceGetUuid
*/
