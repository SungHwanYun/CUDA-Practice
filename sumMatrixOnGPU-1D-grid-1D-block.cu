#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<string.h>

// error check
#define CHECK(call) \
{ \
    const hipError_t error = call;\
    if (error != hipSuccess) { \
        printf("[device] Error: %s %d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(-10*error); \
    } \
}

// Initialize the data pointed to by the pointer ip.
void initialInt(int* ip, const int N) {
    for (int i = 0; i < N; i++) {
        ip[i] = i;
    }
}

// Check if the computation results of the CPU and GPU are the same
void checkResult(float* hostRef, float* gpuRef, const int N) {
    double epsilon = 1.0e-8;
    bool match = 1;
    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("[host] Arrays do not match!\n");
            printf("[host] host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if (match) printf("[host] Arrays match.\n\n");
}

void sumMatrixOnHost(float* A, float* B, float* C, const int nx, const int ny) {
    float* ia = A, * ib = B, * ic = C;
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx; ib += nx; ic += nx;
    }
}

// Initialize the data pointed to by the pointer ip.
#include<time.h>
void initialData(float* ip, const int N) {
    // generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0; i < N; i++) {
        ip[i] = (float)(rand() & 0xff) / 10.0f;
    }
}

__global__ void sumMatrixOnGPU2D2D(float* A, float* B, float* C, const int nx, const int ny) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny) {
        C[idx] = A[idx] + B[idx];
    }
}
__global__ void sumMatrixOnGPU1D1D(float* A, float* B, float* C, const int nx, const int ny) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix < nx) {
        for (int iy = 0; iy < ny; iy++) {
            unsigned int idx = iy * nx + ix;
            C[idx] = A[idx] + B[idx];
        }
    }
}
int main(int argc, char **argv) {
    printf("[host] %s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[host] Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int nx = 1 << 14;
    int ny = 1 << 14;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("[host] Matrix size : nx %d, ny %d\n", nx, ny);

    // malloc host memory
    float* h_A, * h_B, * hostRef, * gpuRef;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);

    // initialize data at host side
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);

    // malloc device global memory
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, nBytes);
    hipMalloc((void**)&d_B, nBytes);
    hipMalloc((void**)&d_C, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // set up execution configuration
    int dimx = 32;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);

    // invoke the kernel
    sumMatrixOnGPU1D1D << <grid, block >> > (d_A, d_B, d_C, nx, ny);
    hipDeviceSynchronize();

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // check device results
    checkResult(hostRef, gpuRef, nxy);
    
    // free host and device memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(hostRef); free(gpuRef);

    // reset device
    hipDeviceReset();
}

/*
output:
C:\coding\Cuda\x64\Debug>nvprof ./Cuda.exe
[host] ./Cuda.exe Starting...
==2304== NVPROF is profiling process 2304, command: ./Cuda.exe
[host] Using Device 0: NVIDIA GeForce MX450
[host] Matrix size : nx 16384, ny 16384
[host] Arrays match.

==2304== Profiling application: ./Cuda.exe
==2304== Warning: 28 API trace records have same start and end timestamps.
This can happen because of short execution duration of CUDA APIs and low timer resolution on the underlying operating system.
==2304== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   49.74%  728.35ms         2  364.18ms  358.11ms  370.24ms  [CUDA memcpy HtoD]
                   26.81%  392.64ms         1  392.64ms  392.64ms  392.64ms  [CUDA memcpy DtoH]
                   23.44%  343.27ms         1  343.27ms  343.27ms  343.27ms  sumMatrixOnGPU1D1D(float*, float*, float*, int, int)
      API calls:   60.08%  1.38027s         3  460.09ms  370.42ms  616.72ms  hipMemcpy
                   14.94%  343.30ms         1  343.30ms  343.30ms  343.30ms  hipDeviceSynchronize
                   13.91%  319.51ms         3  106.50ms  1.9721ms  284.42ms  hipMalloc
                    5.85%  134.45ms         3  44.816ms  9.1093ms  107.47ms  hipFree
                    3.49%  80.164ms         1  80.164ms  80.164ms  80.164ms  hipSetDevice
                    1.68%  38.490ms         1  38.490ms  38.490ms  38.490ms  hipDeviceReset
                    0.05%  1.1568ms         1  1.1568ms  1.1568ms  1.1568ms  hipLaunchKernel
                    0.00%  30.800us         1  30.800us  30.800us  30.800us  hipLibraryUnload
                    0.00%  20.200us       114     177ns       0ns  3.6000us  hipDeviceGetAttribute
                    0.00%  4.0000us         1  4.0000us  4.0000us  4.0000us  hipGetDeviceProperties
                    0.00%  2.1000us         3     700ns     100ns  1.7000us  hipGetDeviceCount
                    0.00%  2.0000us         1  2.0000us  2.0000us  2.0000us  cuModuleGetLoadingMode
                    0.00%  1.9000us         1  1.9000us  1.9000us  1.9000us  hipDeviceTotalMem
                    0.00%  1.0000us         1  1.0000us  1.0000us  1.0000us  hipDeviceGetName
                    0.00%     900ns         2     450ns       0ns     900ns  hipDeviceGet
                    0.00%     300ns         1     300ns     300ns     300ns  cuDeviceGetLuid
                    0.00%     200ns         1     200ns     200ns     200ns  hipDeviceGetUuid
*/
