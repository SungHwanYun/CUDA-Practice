#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<time.h>
#include<string.h>

// If your development environment is Linux, remove the comment below.
// #define LINUX

// error check
#define CHECK(call) \
{ \
    const hipError_t error = call;\
    if (error != hipSuccess) { \
        printf("[device] Error: %s %d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

// Check if the computation results of the CPU and GPU are the same
void checkResult(float* hostRef, float* gpuRef, const int N) {
    double epsilon = 1.0e-8;
    bool match = 1;
    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("[host] Arrays do not match!\n");
            printf("[host] host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if (match) printf("[host] Arrays match.\n\n");
}

// Initialize the data pointed to by the pointer ip.
void initialData(float* ip, const int N) {
    // generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0; i < N; i++) {
        ip[i] = (float)(rand() & 0xff) / 10.0f;
    }
}

// C[i] = A[i] + B[i], 0<= i <= N-1
void sumArraysOnHost(float* A, float* B, float* C, const int N) {
    for (int i = 0; i < N; i++) {
        C[i] = A[i] + B[i];
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, int const N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
    //printf("[device] thread %d : %5.2f + %5.2f = %5.2f\n", i, A[i], B[i], C[i]);
}

double cpuSecond() {
#ifdef LINUX
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tv.tv_sec + (double)tv.tv_usec * 1e-6);
#else 
    return 0.0f;
#endif
}

int main(int argc, char **argv) {
    printf("[host] %s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[host] Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 1 << 24;
    printf("[host] Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);
    float* h_A, * h_B, * hostRef, * gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);

    double iStart, iElaps;

    // initialize data at host side
    iStart = cpuSecond();
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    iElaps = cpuSecond() - iStart;
    memset(hostRef, 0, sizeof(hostRef));
    memset(gpuRef, 0, sizeof(gpuRef));

    // add vector at host side for result checks
    iStart = cpuSecond();
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    iElaps = cpuSecond() - iStart;

    // malloc device global memory
    float* d_A, * d_B, * d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    int iLen = 1 << 10;
    dim3 block(iLen, 1, 1);
    dim3 grid((nElem + block.x - 1) / block.x, 1, 1);

    iStart = cpuSecond();
    sumArraysOnGPU << <grid, block >> > (d_A, d_B, d_C, nElem);
    iElaps = cpuSecond() - iStart;
    printf("[host] sumArraysOnGPU <<<%d, %d>>> Time elapsed %f sec\n", 
        grid.x, block.x, iElaps);

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
}

/*
output:
C:\coding\Cuda\x64\Debug>nvprof ./Cuda.exe
[host] ./Cuda.exe Starting...
==20084== NVPROF is profiling process 20084, command: ./Cuda.exe
[host] Using Device 0: NVIDIA GeForce MX450
[host] Vector size 16777216
[host] sumArraysOnGPU <<<16384, 1024>>> Time elapsed 0.000000 sec
[host] Arrays match.

==20084== Profiling application: ./Cuda.exe
==20084== Warning: 36 API trace records have same start and end timestamps.
This can happen because of short execution duration of CUDA APIs and low timer resolution on the underlying operating system.
==20084== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   62.71%  43.880ms         2  21.940ms  21.776ms  22.104ms  [CUDA memcpy HtoD]
                   31.41%  21.979ms         1  21.979ms  21.979ms  21.979ms  [CUDA memcpy DtoH]
                    5.88%  4.1108ms         1  4.1108ms  4.1108ms  4.1108ms  sumArraysOnGPU(float*, float*, float*, int)
      API calls:   38.87%  138.87ms         1  138.87ms  138.87ms  138.87ms  hipSetDevice
                   26.96%  96.324ms         1  96.324ms  96.324ms  96.324ms  hipLaunchKernel
                   20.71%  73.983ms         3  24.661ms  22.211ms  26.619ms  hipMemcpy
                    9.82%  35.090ms         1  35.090ms  35.090ms  35.090ms  hipDevicePrimaryCtxRelease
                    2.95%  10.536ms         3  3.5121ms  166.00us  9.6766ms  hipMalloc
                    0.56%  1.9937ms         3  664.57us  398.20us  1.1074ms  hipFree
                    0.06%  217.50us         1  217.50us  217.50us  217.50us  cuModuleGetLoadingMode
                    0.06%  211.40us       114  1.8540us       0ns  196.00us  hipDeviceGetAttribute
                    0.01%  46.400us         1  46.400us  46.400us  46.400us  hipLibraryUnload
                    0.00%  5.9000us         1  5.9000us  5.9000us  5.9000us  hipGetDeviceProperties
                    0.00%  2.4000us         3     800ns     100ns  2.0000us  hipGetDeviceCount
                    0.00%  1.7000us         1  1.7000us  1.7000us  1.7000us  hipDeviceTotalMem
                    0.00%  1.1000us         2     550ns     100ns  1.0000us  hipDeviceGet
                    0.00%  1.0000us         1  1.0000us  1.0000us  1.0000us  hipDeviceGetName
                    0.00%     500ns         1     500ns     500ns     500ns  cuDeviceGetLuid
*/
