#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<string.h>

// error check
#define CHECK(call) \
{ \
    const hipError_t error = call;\
    if (error != hipSuccess) { \
        printf("[device] Error: %s %d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(-10*error); \
    } \
}

// Check if the computation results of the CPU and GPU are the same
void checkResult(float* hostRef, float* gpuRef, const int N) {
    double epsilon = 1.0e-8;
    bool match = 1;
    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("[host] Arrays do not match!\n");
            printf("[host] host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if (match) printf("[host] Arrays match.\n\n");
}

// Initialize the data pointed to by the pointer ip.
#include<time.h>
void initialData(int* ip, const int N) {
    // generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0; i < N; i++) {
        ip[i] = (int)(rand() & 0xff);
    }
}

int recursiveReduce(int* data, const int size) {
    // terminate check
    if (size == 1) return data[0];

    // renew the stride
    const int stride = size / 2;

    // in-place reduction
    for (int i = 0; i < stride; i++) {
        data[i] += data[i + stride];
    }
    return recursiveReduce(data, stride);
}
__global__ void warmup(int* g_idata, int* g_odata, const int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block
        __syncthreads();
    }
    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}
__global__ void reduceNeighbored(int* g_idata, int* g_odata, const int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block
        __syncthreads();
    }
    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}
__global__ void reduceNeighboredLess(int* g_idata, int* g_odata, const int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        // convert tid into local array index
        int index = 2 * stride * tid;
        if (index < blockDim.x) {
            idata[index] += idata[index + stride];
        }

        // synchronize within block
        __syncthreads();
    }
    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}
__global__ void reduceInterleaved(int* g_idata, int* g_odata, const int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block
        __syncthreads();
    }
    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}
__global__ void reduceInterleavedUnrolling2(int* g_idata, int* g_odata, const int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 2;

    // boundary check
    if (idx + blockDim.x >= n) return;
    g_idata[idx] += g_idata[idx + blockDim.x];
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block
        __syncthreads();
    }
    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}
__global__ void reduceInterleavedUnrolling4(int* g_idata, int* g_odata, const int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 4;

    // boundary check
    if (idx + blockDim.x * 3 >= n) return;
    int a1 = g_idata[idx];
    int a2 = g_idata[idx + blockDim.x];
    int a3 = g_idata[idx + blockDim.x * 2];
    int a4 = g_idata[idx + blockDim.x * 3];
    g_idata[idx] = a1 + a2 + a3 + a4;
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block
        __syncthreads();
    }
    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}
__global__ void reduceInterleavedUnrolling8(int* g_idata, int* g_odata, const int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 8;

    // boundary check
    if (idx + blockDim.x * 7 >= n) return;
    int a1 = g_idata[idx];
    int a2 = g_idata[idx + blockDim.x];
    int a3 = g_idata[idx + blockDim.x * 2];
    int a4 = g_idata[idx + blockDim.x * 3];
    int a5 = g_idata[idx + blockDim.x * 4];
    int a6 = g_idata[idx + blockDim.x * 5];
    int a7 = g_idata[idx + blockDim.x * 6];
    int a8 = g_idata[idx + blockDim.x * 7];
    g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block
        __syncthreads();
    }
    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}
__global__ void reduceInterleavedUnrollingWarp8(int* g_idata, int* g_odata, const int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 8;

    // boundary check
    if (idx + blockDim.x * 7 >= n) return;
    int a1 = g_idata[idx];
    int a2 = g_idata[idx + blockDim.x];
    int a3 = g_idata[idx + blockDim.x * 2];
    int a4 = g_idata[idx + blockDim.x * 3];
    int a5 = g_idata[idx + blockDim.x * 4];
    int a6 = g_idata[idx + blockDim.x * 5];
    int a7 = g_idata[idx + blockDim.x * 6];
    int a8 = g_idata[idx + blockDim.x * 7];
    g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32) {
        volatile int* vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}
__global__ void reduceInterleavedCompleteUnrollingWarp8(int* g_idata, int* g_odata, const int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 8;

    // boundary check
    if (idx + blockDim.x * 7 >= n) return;
    int a1 = g_idata[idx];
    int a2 = g_idata[idx + blockDim.x];
    int a3 = g_idata[idx + blockDim.x * 2];
    int a4 = g_idata[idx + blockDim.x * 3];
    int a5 = g_idata[idx + blockDim.x * 4];
    int a6 = g_idata[idx + blockDim.x * 5];
    int a7 = g_idata[idx + blockDim.x * 6];
    int a8 = g_idata[idx + blockDim.x * 7];
    g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int* vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}
int main(int argc, char** argv) {
    printf("[host] %s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[host] Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size
    int size = 1 << 24; 
    size_t nBytes = size * sizeof(int);
    printf("[host] Data size : %d\n", size);

    // set up execution configuration
    int blocksize = 512;
    if (argc > 1) {
        blocksize = atoi(argv[1]);
    }
    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    printf("[host] Execution configure : grid(%d, %d), block(%d, %d)\n", 
        grid.x, grid.y, block.x, block.y);

    // allocate host memory
    int* h_idata = (int*)malloc(nBytes);
    int* h_odata = (int*)malloc(grid.x*sizeof(int));
    int* tmp = (int*)malloc(nBytes);

    // initialize the array
    initialData(h_idata, size);
    memcpy(tmp, h_idata, nBytes);

    // allocate device memory
    int* d_idata, * d_odata;
    hipMalloc((void**)&d_idata, nBytes);
    hipMalloc((void**)&d_odata, grid.x*sizeof(int));

    // cpu reduction
    int cpu_sum = recursiveReduce(tmp, size);

    // warmup kernel
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    warmup << <grid, block >> > (d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    int gpu_sum = 0;
    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];
    printf("[host] gpu warmup : grid(%d, %d), block(%d, %d), cpu_sum=%d, gpu_sum=%d\n",
        grid.x, grid.y, block.x, block.y, cpu_sum, gpu_sum);

    // kernel 1: reduceNeighbored
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reduceNeighbored << <grid, block >> > (d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];
    printf("[host] gpu reduceNeighbored : grid(%d, %d), block(%d, %d), cpu_sum=%d, gpu_sum=%d\n",
        grid.x, grid.y, block.x, block.y, cpu_sum, gpu_sum);

    // kernel 2: reduceNeighboredLess
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reduceNeighboredLess << <grid, block >> > (d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];
    printf("[host] gpu reduceNeighboredLess : grid(%d, %d), block(%d, %d), cpu_sum=%d, gpu_sum=%d\n",
        grid.x, grid.y, block.x, block.y, cpu_sum, gpu_sum);

    // kernel 3: reduceInterleaved
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reduceInterleaved << <grid, block>> > (d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];
    printf("[host] gpu reduceInterleaved : grid(%d, %d), block(%d, %d), cpu_sum=%d, gpu_sum=%d\n",
        grid.x, grid.y, block.x, block.y, cpu_sum, gpu_sum);

    // kernel 4: reduceInterleavedUnrolling2
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reduceInterleavedUnrolling2 <<< grid.x / 2, block >>> (d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 2; i++) gpu_sum += h_odata[i];
    printf("[host] gpu reduceInterleavedUnrolling2 : grid(%d, %d), block(%d, %d), cpu_sum=%d, gpu_sum=%d\n",
        grid.x / 2, grid.y, block.x, block.y, cpu_sum, gpu_sum);

    // kernel 5: reduceInterleavedUnrolling4
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reduceInterleavedUnrolling4 << < grid.x / 4, block >> > (d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 4; i++) gpu_sum += h_odata[i];
    printf("[host] gpu reduceInterleavedUnrolling4 : grid(%d, %d), block(%d, %d), cpu_sum=%d, gpu_sum=%d\n",
        grid.x / 4, grid.y, block.x, block.y, cpu_sum, gpu_sum);

    // kernel 6: reduceInterleavedUnrolling8
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reduceInterleavedUnrolling8 << < grid.x / 8, block >> > (d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++) gpu_sum += h_odata[i];
    printf("[host] gpu reduceInterleavedUnrolling8 : grid(%d, %d), block(%d, %d), cpu_sum=%d, gpu_sum=%d\n",
        grid.x / 8, grid.y, block.x, block.y, cpu_sum, gpu_sum);

    // kernel 7: reduceInterleavedUnrollingWarp8
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reduceInterleavedUnrollingWarp8 << < grid.x / 8, block >> > (d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++) gpu_sum += h_odata[i];
    printf("[host] gpu reduceInterleavedUnrollingWarp8 : grid(%d, %d), block(%d, %d), cpu_sum=%d, gpu_sum=%d\n",
        grid.x / 8, grid.y, block.x, block.y, cpu_sum, gpu_sum);

    // kernel 8: reduceInterleavedCompleteUnrollingWarp8
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reduceInterleavedCompleteUnrollingWarp8 << < grid.x / 8, block >> > (d_idata, d_odata, size);
    hipDeviceSynchronize();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++) gpu_sum += h_odata[i];
    printf("[host] gpu reduceInterleavedCompleteUnrollingWarp8 : grid(%d, %d), block(%d, %d), cpu_sum=%d, gpu_sum=%d\n",
        grid.x / 8, grid.y, block.x, block.y, cpu_sum, gpu_sum);

    // free host memory
    free(h_idata); free(h_odata); free(tmp);

    // free device memory
    hipFree(d_idata); hipFree(d_odata);

    // reset device
    hipDeviceReset();
}

/*
output:
C:\coding\Cuda\x64\Debug>nvprof ./Cuda
[host] ./Cuda Starting...
==3788== NVPROF is profiling process 3788, command: ./Cuda
[host] Using Device 0: NVIDIA GeForce MX450
[host] Data size : 16777216
[host] Execution configure : grid(32768, 1), block(512, 1)
[host] gpu warmup : grid(32768, 1), block(512, 1), cpu_sum=2139095040, gpu_sum=2139095040
[host] gpu reduceNeighbored : grid(32768, 1), block(512, 1), cpu_sum=2139095040, gpu_sum=2139095040
[host] gpu reduceNeighboredLess : grid(32768, 1), block(512, 1), cpu_sum=2139095040, gpu_sum=2139095040
[host] gpu reduceInterleaved : grid(32768, 1), block(512, 1), cpu_sum=2139095040, gpu_sum=2139095040
[host] gpu reduceInterleavedUnrolling2 : grid(16384, 1), block(512, 1), cpu_sum=2139095040, gpu_sum=2139095040
[host] gpu reduceInterleavedUnrolling4 : grid(8192, 1), block(512, 1), cpu_sum=2139095040, gpu_sum=2139095040
[host] gpu reduceInterleavedUnrolling8 : grid(4096, 1), block(512, 1), cpu_sum=2139095040, gpu_sum=2139095040
[host] gpu reduceInterleavedUnrollingWarp8 : grid(4096, 1), block(512, 1), cpu_sum=2139095040, gpu_sum=2139095040
[host] gpu reduceInterleavedCompleteUnrollingWarp8 : grid(4096, 1), block(512, 1), cpu_sum=2139095040, gpu_sum=2139095040
==3788== Profiling application: ./Cuda
==3788== Warning: 33 API trace records have same start and end timestamps.
This can happen because of short execution duration of CUDA APIs and low timer resolution on the underlying operating system.
==3788== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   77.55%  206.32ms         9  22.925ms  21.951ms  23.678ms  [CUDA memcpy HtoD]
                    5.20%  13.833ms         1  13.833ms  13.833ms  13.833ms  warmup(int*, int*, int)
                    5.20%  13.827ms         1  13.827ms  13.827ms  13.827ms  reduceNeighbored(int*, int*, int)
                    3.66%  9.7498ms         1  9.7498ms  9.7498ms  9.7498ms  reduceNeighboredLess(int*, int*, int)
                    3.52%  9.3520ms         1  9.3520ms  9.3520ms  9.3520ms  reduceInterleaved(int*, int*, int)
                    1.89%  5.0383ms         1  5.0383ms  5.0383ms  5.0383ms  reduceInterleavedUnrolling2(int*, int*, int)
                    0.87%  2.3141ms         1  2.3141ms  2.3141ms  2.3141ms  reduceInterleavedUnrolling4(int*, int*, int)
                    0.69%  1.8249ms         1  1.8249ms  1.8249ms  1.8249ms  reduceInterleavedUnrollingWarp8(int*, int*, int)
                    0.67%  1.7912ms         1  1.7912ms  1.7912ms  1.7912ms  reduceInterleavedCompleteUnrollingWarp8(int*, int*, int)
                    0.61%  1.6239ms         1  1.6239ms  1.6239ms  1.6239ms  reduceInterleavedUnrolling8(int*, int*, int)
                    0.14%  361.57us         9  40.174us  38.400us  42.144us  [CUDA memcpy DtoH]
      API calls:   44.33%  205.35ms        18  11.408ms  117.70us  23.422ms  hipMemcpy
                   25.37%  117.53ms         1  117.53ms  117.53ms  117.53ms  hipSetDevice
                   14.00%  64.847ms        18  3.6026ms  474.00us  13.912ms  hipDeviceSynchronize
                    8.51%  39.410ms         9  4.3789ms  49.400us  38.890ms  hipLaunchKernel
                    7.48%  34.652ms         1  34.652ms  34.652ms  34.652ms  hipDeviceReset
                    0.22%  1.0186ms         2  509.30us  265.00us  753.60us  hipFree
                    0.08%  360.70us         2  180.35us  83.300us  277.40us  hipMalloc
                    0.01%  58.900us         1  58.900us  58.900us  58.900us  hipLibraryUnload
                    0.00%  19.400us       114     170ns       0ns  2.6000us  hipDeviceGetAttribute
                    0.00%  3.8000us         1  3.8000us  3.8000us  3.8000us  hipGetDeviceProperties
                    0.00%  2.3000us         3     766ns       0ns  2.1000us  hipGetDeviceCount
                    0.00%  1.8000us         1  1.8000us  1.8000us  1.8000us  cuModuleGetLoadingMode
                    0.00%  1.7000us         1  1.7000us  1.7000us  1.7000us  hipDeviceTotalMem
                    0.00%  1.0000us         2     500ns     100ns     900ns  hipDeviceGet
                    0.00%     700ns         1     700ns     700ns     700ns  hipDeviceGetName
                    0.00%     300ns         1     300ns     300ns     300ns  cuDeviceGetLuid
                    0.00%     200ns         1     200ns     200ns     200ns  hipDeviceGetUuid
*/
